#include "hip/hip_runtime.h"
#include "GlobalData/Types/TypesHandler.cuh"
#include "GlobalData/Types/TypesFactory.cuh"
#include "GlobalData/Types/Types/Types.cuh"

namespace uammd{
namespace structured{
namespace Types{

    struct Basic_{

        template<typename T>
        static void loadType(std::map<std::string,std::map<std::string,real>>& nameToData,
                             std::map<std::string,T>& typeData){

            std::string name = typeData.at("name");

            nameToData[name]["mass"]   = real(typeData.at("mass"));
            nameToData[name]["radius"] = real(typeData.at("radius"));
            nameToData[name]["charge"] = real(typeData.at("charge"));

            System::log<System::MESSAGE>("[Basic] Loaded type %s, mass: %f, radius: %f, charge: %f",
                                         name.c_str(), nameToData[name]["mass"],
                                         nameToData[name]["radius"], nameToData[name]["charge"]);
        }

        static void loadTypesIntoParticleData(std::shared_ptr<ParticleData> pd,
                                              std::map<int,std::string>&    idToName,
                                              std::map<std::string,std::map<std::string,real>>& nameToData){

            int N = pd->getNumParticles();

            auto pos     = pd->getPos(access::location::cpu,access::mode::read);

            //Check if mass,radius or charge are already defined

            bool massDefined   = pd->isMassAllocated();
            bool radiusDefined = pd->isRadiusAllocated();
            bool chargeDefined = pd->isChargeAllocated();

            if(massDefined){
                System::log<System::WARNING>("[Basic] Mass is already defined, ignoring mass from type");
            }
            if(radiusDefined){
                System::log<System::WARNING>("[Basic] Radius is already defined, ignoring radius from type");
            }
            if(chargeDefined){
                System::log<System::WARNING>("[Basic] Charge is already defined, ignoring charge from type");
            }

            auto mass    = pd->getMass(access::location::cpu,   access::mode::write);
            auto radius  = pd->getRadius(access::location::cpu, access::mode::write);
            auto charge  = pd->getCharge(access::location::cpu, access::mode::write);

            for(int i = 0; i < N; i++){

                std::string name = idToName.at(int(pos[i].w));

                if(!massDefined)  { mass[i]   = nameToData[name]["mass"]; }
                if(!radiusDefined){ radius[i] = nameToData[name]["radius"]; }
                if(!chargeDefined){ charge[i] = nameToData[name]["charge"]; }

                System::log<System::DEBUG1>("[Basic] Loading type for particle %d, mass: %f, radius: %f, charge: %f",i,mass[i],radius[i],charge[i]);
            }
        }
    };

    using Basic = Types_<Basic_>;

}}}

REGISTER_TYPES(
    Types,Basic,
    uammd::structured::Types::Basic
)
