#include "hip/hip_runtime.h"
#include "System/ExtendedSystem.cuh"
#include "GlobalData/GlobalData.cuh"
#include "ParticleData/ExtendedParticleData.cuh"
#include "ParticleData/ParticleGroup.cuh"
#include "ParticleGroup/ParticleGroupUtils.cuh"

#include "DataStructures/VerletConditionalListSet/VerletConditionalListSetBase.cuh"
#include "DataStructures/VerletConditionalListSet/VerletConditionalListSetFactory.cuh"

#include "DataStructures/VerletConditionalListSet/VerletConditionalListSetLoaders.cuh"
#include "DataStructures/VerletConditionalListSet/VerletConditionalListSetUtils.cuh"

namespace uammd{
namespace structured{
namespace VerletConditionalListSetLoaders{

    std::shared_ptr<uammd::structured::VerletConditionalListSetBase>
    loadVerletConditionalListSet(std::shared_ptr<ExtendedSystem> sys,
                                 std::shared_ptr<GlobalData>    gd,
                                 std::map<std::string,std::shared_ptr<ParticleGroup>> groups,
                                 std::vector<std::string>       path){


        DataEntry data = sys->getInput()->getDataEntry(path);
        //Check data type is VerletConditionalListSet
        if(data.getType() != "VerletConditionalListSet"){
            System::log<System::CRITICAL>("[VerletConditionalListSetLoader] (%s) Data type is not VerletConditionalListSet!",path.back().c_str());
        }

        std::shared_ptr<ParticleGroup> pg = GroupUtils::getParticleGroupFromGroupsList(groups,data,"All");

        //////////////////////////////////////////////////////////////////////////////

        std::string condition = data.getSubType();

        return VerletConditionalListSetFactory::getInstance().createVerletConditionalListSet(condition,
                                                                                             gd,pg,
                                                                                             data,
                                                                                             path.back());

    }
}}}
