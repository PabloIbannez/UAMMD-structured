#include "System/ExtendedSystem.cuh"
#include "GlobalData/GlobalData.cuh"
#include "ParticleData/ExtendedParticleData.cuh"
#include "ParticleData/ParticleGroup.cuh"

#include "Interactor/Bonds/BondsInteractor.cuh"
#include "Interactor/Bonds/Bond2/Bond2.cuh"
#include "Interactor/InteractorFactory.cuh"

#include "Interactor/BasicPotentials/RotationalAlignmentPotential.cuh"

namespace uammd{
namespace structured{
namespace Potentials{
namespace Bond2{

    struct RAP_{

        struct ComputationalData{
            real4* pos;
            real4* dir;

            Box    box;
        };

        //Potential parameters
        struct StorageData{};

        struct BondParameters{
            real K;
            Quat R;// Rotation encoded as a quaternion
        };

        //Computational data getter

        static __host__ ComputationalData getComputationalData(std::shared_ptr<GlobalData>    gd,
                                                               std::shared_ptr<ParticleGroup> pg,
                                                               const StorageData&  storage,
                                                               const Computables& computables,
                                                               const hipStream_t& st){

            ComputationalData computational;

            std::shared_ptr<ParticleData> pd = pg->getParticleData();

            computational.pos = pd->getPos(access::location::gpu, access::mode::read).raw();
            computational.dir = pd->getDir(access::location::gpu, access::mode::read).raw();

            computational.box = gd->getEnsemble()->getBox();

            return computational;
        }

        //Storage data reader

        static __host__ StorageData getStorageData(std::shared_ptr<GlobalData>           gd,
                                                   std::shared_ptr<ParticleGroup>        pg,
                                                   DataEntry& data){

            StorageData storage;

            return storage;
        }

        //Bond parameters reader

        template<typename T>
        static __host__ BondParameters processBondParameters(std::shared_ptr<GlobalData> gd,
                                                             std::map<std::string,T>& bondParametersMap){

            BondParameters param;
            param.K   = bondParametersMap.at("K");
            real4 R   = bondParametersMap.at("R");
            param.R   = Quat(R);
            return param;
        }

        //Energy and force definition
        static inline __device__ real energy(int index_i, int index_j,
                                             int currentParticleIndex,
                                             const ComputationalData &computational,
                                             const BondParameters &bondParam){


            const tensor3 A = MatrixOperations::quat2mat(computational.dir[index_i]);
            const tensor3 B = MatrixOperations::quat2mat(computational.dir[index_j]);

            real    K = bondParam.K;
            tensor3 R = MatrixOperations::quat2mat(bondParam.R);

            if(index_j == currentParticleIndex){
                R = R.transpose();
            }

            real e = BasicPotentials::RAP::energy(A, B, R);

            return e;
        }

        static inline __device__ ForceTorque forceTorque(int index_i, int index_j,
                                                         int currentParticleIndex,
                                                         const ComputationalData &computational,
                                                         const BondParameters &bondParam){
            ForceTorque forceTorque;
            forceTorque.force = make_real4(0.0);

            const tensor3 A = MatrixOperations::quat2mat(computational.dir[index_i]);
            const tensor3 B = MatrixOperations::quat2mat(computational.dir[index_j]);

            real    K = bondParam.K;
            tensor3 R = MatrixOperations::quat2mat(bondParam.R);

            if(index_j == currentParticleIndex){
                R = R.transpose();
            }

            real3 t = -K*BasicPotentials::RAP::torque(A, B, R)/real(4.0);
            forceTorque.torque = make_real4(t, 0.0);

            return forceTorque;
        }


    };

    using RAP = Bond2Torque_<RAP_>;

}}}}

REGISTER_BOND_INTERACTOR(
    Bond2,RAP,
    uammd::structured::Interactor::BondsInteractor<uammd::structured::Potentials::Bond2::RAP>
)
