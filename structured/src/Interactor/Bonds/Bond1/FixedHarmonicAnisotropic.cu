#include "System/ExtendedSystem.cuh"
#include "GlobalData/GlobalData.cuh"
#include "ParticleData/ExtendedParticleData.cuh"
#include "ParticleData/ParticleGroup.cuh"

#include "Interactor/Bonds/BondsInteractor.cuh"
#include "Interactor/Bonds/Bond1/Bond1.cuh"
#include "Interactor/InteractorFactory.cuh"

#include "Interactor/BasicPotentials/Harmonic.cuh"

namespace uammd{
namespace structured{
namespace Potentials{
namespace Bond1{

    struct FixedHarmonicAnisotropic_{

        struct ComputationalData{
            real4* pos;
            Box    box;
        };

        //Potential parameters

        struct StorageData{};

        struct BondParameters{
            real3 K;
            real3 r0;
            real3 pos;
        };

        //Computational data getter

        static __host__ ComputationalData getComputationalData(std::shared_ptr<GlobalData>    gd,
                                                               std::shared_ptr<ParticleGroup> pg,
                                                               const StorageData&  storage,
                                                               const Computables& computables,
                                                               const hipStream_t& st){

            ComputationalData computational;

            std::shared_ptr<ParticleData> pd = pg->getParticleData();

            computational.pos = pd->getPos(access::location::gpu, access::mode::read).raw();
            computational.box = gd->getEnsemble()->getBox();

            return computational;
        }

        //Storage data reader

        static __host__ StorageData getStorageData(std::shared_ptr<GlobalData>    gd,
                                                   std::shared_ptr<ParticleGroup> pg,
                                                   DataEntry& data){

            StorageData storage;
            return storage;
        }

        //Bond parameters reader

        template<typename T>
        static __host__ BondParameters processBondParameters(std::shared_ptr<GlobalData> gd,
                                                             std::map<std::string,T>& bondParametersMap){

            BondParameters param;

            param.K    = bondParametersMap.at("K");
            param.r0   = bondParametersMap.at("r0");
            param.pos  = bondParametersMap.at("position");

            return param;
        }

        //Energy and force definition

        static inline __device__ real3 force(int index_i,
                                             int currentParticleIndex,
                                             const ComputationalData &computational,
                                             const BondParameters &bondParam){

            const real3 posi = make_real3(computational.pos[index_i]);
            const real3 posj = bondParam.pos;
            const real3 rij  = computational.box.apply_pbc(posj-posi);

            const real3 K   = bondParam.K;
            const real3 r0  = bondParam.r0;

            return BasicPotentials::HarmonicAnisotropic::force(rij,K,r0);
        }

        static inline __device__ real energy(int index_i,
                                             int currentParticleIndex,
                                             const ComputationalData &computational,
                                             const BondParameters &bondParam){

            const real3 posi = make_real3(computational.pos[index_i]);
            const real3 posj = bondParam.pos;
            const real3 rij  = computational.box.apply_pbc(posj-posi);

            const real3 K   = bondParam.K;
            const real3 r0  = bondParam.r0;

            const real e = BasicPotentials::HarmonicAnisotropic::energy(rij,K,r0);

            return e;
        }

      static inline __device__ tensor3 hessian(int index_i,
					       int currentParticleIndex,
					       const ComputationalData &computational,
					       const BondParameters &bondParam){

	tensor3 H = tensor3();
	H.xx = bondParam.K.x; H.yy = bondParam.K.y; H.zz = bondParam.K.z;
	return H;
        }

    };

    using FixedHarmonicAnisotropic = Bond1Hessian_<FixedHarmonicAnisotropic_>;

}}}}

REGISTER_BOND_INTERACTOR(
    Bond1,FixedHarmonicAnisotropic,
    uammd::structured::Interactor::BondsInteractor<uammd::structured::Potentials::Bond1::FixedHarmonicAnisotropic>
)
