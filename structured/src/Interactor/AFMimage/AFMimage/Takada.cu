#include "System/ExtendedSystem.cuh"
#include "GlobalData/GlobalData.cuh"
#include "ParticleData/ExtendedParticleData.cuh"
#include "ParticleData/ParticleGroup.cuh"

#include "Interactor/AFMimage/AFMimageInteractor.cuh"
#include "Interactor/AFMimage/AFMimage/AFMimage.cuh"
#include "Interactor/InteractorFactory.cuh"

namespace uammd{
namespace structured{
namespace Potentials{
namespace AFMimage{

    struct Takada_{

        struct ComputationalData {};

        struct StorageData {};

        //Computational data getter
        static ComputationalData getComputationalData(std::shared_ptr<GlobalData>    gd,
                                                      std::shared_ptr<ParticleGroup> pg,
                                                      const StorageData&  storage,
                                                      const Computables& comp,
                                                      const hipStream_t& st){
            ComputationalData computational;
            return computational;
        }

        //Storage data reader
        static StorageData getStorageData(std::shared_ptr<GlobalData>    gd,
                                          std::shared_ptr<ParticleGroup> pg,
                                          DataEntry& data){

            StorageData storage;
            return storage;
        }

        //Energy and force definition

        static inline __device__ real energy(int index,
                                             int tipIndex,
                                             const ComputationalData &computational){
            const real e = real(0.0);
            return e;
        }

        static inline __device__ real3 force(int index,
                                             int tipIndex,
                                             const ComputationalData &computational){
            real3 f = make_real3(0.0);
            return f;

        }

    };

    using Takada = AFMimage_<Takada_>;

}}}}

REGISTER_AFM_IMAGE_INTERACTOR(
    AFMimage,Takada,
    uammd::structured::Interactor::AFMimageInteractor<uammd::structured::Potentials::AFMimage::Takada>
)
