#include "hip/hip_runtime.h"
#include "System/ExtendedSystem.cuh"
#include "GlobalData/GlobalData.cuh"
#include "ParticleData/ExtendedParticleData.cuh"
#include "ParticleData/ParticleGroup.cuh"
#include "ParticleGroup/ParticleGroupUtils.cuh"

#include "Integrator/IntegratorBase.cuh"
#include "Integrator/IntegratorFactory.cuh"
#include "Integrator/IntegratorUtils.cuh"

#include "Integrator/BDHI/BDHI_EulerMaruyama.cuh"
#include "Integrator/BDHI/BDHI_Lanczos.cuh"

namespace uammd{
namespace structured{
namespace Integrator{
namespace NVT{
namespace BDHIOpenBoundary{

	class Lanczos : public IntegratorBaseNVT{

		private:

			using BDHI = BDHI::EulerMaruyama<BDHI::Lanczos>;

			std::unique_ptr<BDHI> bdhi;
			bool firstStep = true;

		public:

      Lanczos(std::shared_ptr<GlobalData>           gd,
              std::shared_ptr<ParticleGroup>        pg,
              DataEntry& data,
              std::string name):IntegratorBaseNVT(gd,pg,data,name){

				int batchNumber = GroupUtils::BatchGroupNumber(pg);
				if(batchNumber > 1){
					System::log<System::CRITICAL>("[BDHI] This integrator can not handle more than one batch.");
				}

				BDHI::Parameters bdhiParameters;

				bdhiParameters.dt = this->dt;

				bdhiParameters.temperature = this->kBT;
				bdhiParameters.viscosity = data.getParameter<real>("viscosity");

				bdhiParameters.hydrodynamicRadius = data.getParameter<real>("hydrodynamicRadius",-1.0);

				bdhiParameters.tolerance = data.getParameter<real>("tolerance",1e-3);

				System::log<System::MESSAGE>("[BDHI] Viscosity: ",bdhiParameters.viscosity);

				if(bdhiParameters.hydrodynamicRadius < 0.0){
					System::log<System::MESSAGE>("[BDHI] Hydrodynamic radius not set, using particle radius");
				} else {
					System::log<System::MESSAGE>("[BDHI] Hydrodynamic radius: %f",bdhiParameters.hydrodynamicRadius);
				}

				System::log<System::MESSAGE>("[BDHI] Tolerance: %f",bdhiParameters.tolerance);

				bdhi = std::make_unique<BDHI>(pg,bdhiParameters);

			}

			void forwardTime() override {

				if(firstStep){
					//Load all interactors into bdhi
					for(auto& interactor : this->getInteractors()){
						bdhi->addInteractor(interactor);
					}

					//Load all updatables into bdhi
					for(auto& updatable : this->getUpdatables()){
						bdhi->addUpdatable(updatable);
					}

					firstStep = false;
				}

				bdhi->forwardTime();
        this->gd->getFundamental()->setCurrentStep(this->gd->getFundamental()->getCurrentStep()+1);
        this->gd->getFundamental()->setSimulationTime(this->gd->getFundamental()->getSimulationTime()+this->dt);
			}

	};

}}}}}

REGISTER_INTEGRATOR(
    BDHIOpenBoundary,Lanczos,
    uammd::structured::Integrator::NVT::BDHIOpenBoundary::Lanczos
)
